#include "hip/hip_runtime.h"
#include "stdlib.h"
#include "stdio.h"
#include <iostream>


#include <ctime>

using namespace std;

__global__ void sum_cuda(int *res, int *mas1, int *mas2, int N, int M)
{
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	int j = threadIdx.y + blockIdx.y * blockDim.y;
	int tx = i * N + j;
	res[tx] = mas1[tx] + mas2[tx];
}

void init(int *mas, int size);

void sum(int *res, int *mas1, int *mas2, int N, int M);

void print(int *mas, int N, int M);

int main()
{
	int N = 4, M = 4, dimN = 2, dimM = 2;

	int mas1[N * M];
	init(mas1, N * M);

	int mas2[N * M];
	init(mas2, N*M);

	int *res = new int[N * M];

	int *cudaMas1;
	int *cudaMas2;
	int *cudaRes;

	float time = 0;

	hipMalloc((void**)&cudaMas1, sizeof(int) * N * M);
	hipMalloc((void**)&cudaMas2, sizeof(int) * N * M);
	hipMalloc((void**)&cudaRes,  sizeof(int) * N * M);

	hipMemcpy(cudaMas1, mas1, sizeof(int) * N * M, hipMemcpyHostToDevice);
	hipMemcpy(cudaMas2, mas2, sizeof(int) * N * M, hipMemcpyHostToDevice);
	hipMemcpy(cudaRes,  res,  sizeof(int) * N * M, hipMemcpyHostToDevice);

	hipEvent_t start, end;
	hipEventCreate(&start);
	hipEventCreate(&end);

	print(mas1, N, M);
	print(mas2, N, M);

	hipEventRecord(start);
	sum(res, mas1, mas2, N, M);
	hipEventRecord(end);
	hipEventSynchronize(end);	

	print(res, N, M);
	hipEventElapsedTime(&time, start, end);
	cout << "Последовательно " << time << endl;

	dim3 blocks(N / dimN, M / dimM);
	dim3 threads(dimN, dimM);

	hipEventRecord(start);
	sum_cuda<<< blocks, threads >>>(cudaRes, cudaMas1, cudaMas2, N, M);
	hipDeviceSynchronize();
	hipEventRecord(end);

	hipMemcpy(res, cudaRes, sizeof(int) * N * M, hipMemcpyDeviceToHost);

	print(res, N, M);
 	hipEventElapsedTime(&time, start, end);
   	cout << "Параллельно " << time << endl;

	hipFree(cudaMas1);
	hipFree(cudaMas2);
	hipFree(cudaRes);

	return 0;
}


void sum(int *res, int *mas1, int *mas2, int N, int M)
{
	for (int i = 0; i < N; i++){
		for (int j = 0; j < M; j++)
			res[i*N + j] = mas1[i*N + j] + mas2[i*N + j];
	}
}

void print(int *mas, int N, int M)
{
	for (int i = 0; i < N; i++){
		for (int j = 0; j < M; j++)
			cout << mas[i*N + j] << " ";
		cout << endl;
	}
	cout << endl;
}

void init(int *mas, int size)
{
	srand( time(0));
	for (int i = 0; i < size; i++)
	{
		mas[i] = rand() % 1000;
	}
} 
